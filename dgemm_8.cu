#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#define BLOCK_WIDTH 32
#define TAILLE 4096

#define gettime(t) clock_gettime(CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9*(double)t.tv_nsec)
/** return time in second
*/
double get_elapsedtime(void)
{
  struct timespec st;
  int err = gettime(&st);
  if (err !=0) return 0;
  return (double)st.tv_sec + get_sub_seconde(st);
}

void init(double* A, double* B, double* C, int size)
{
  int i = 0, j = 0;

  srand(2019);

  for(i = 0; i < size; i++)
  {
    for(j = 0; j < size; j++)
    {
      A[i * size + j] = rand();
      B[i * size + j] = rand();
      C[i * size + j] = 0.0;
    }
  }
}

void mult(double* A, double* B, double* C, int size)
{
  int i = 0, j = 0, k = 0;

  for(i = 0; i < size; i++)
  {
    for(j = 0; j < size; j++)
    {
      double sum = 0.;
      for(k = 0; k < size; k++)
      {
        sum += A[i * size + k] * B[k* size + j];
      }
      C[i * size + j] = sum;
    }
  }
}

// QUESTION 4
__global__
void MulMatrixKernel(double* A, double* B, double* C, int N)
{
  // QUESTION 6
  int col    = threadIdx.x + blockDim.x * blockIdx.x;
  int line  = threadIdx.y + blockDim.y * blockIdx.y;
  // FIN QUESTION 6

  // QUESTION 7
  if((col < N) && (line < N))
  {
    double val = 0.0f;
    for(int k = 0; k < N; k++)
    {
      val += A[line * N + k] * B[k * N + col];
    }
    C[line * N + col] = val;
  }
  // FIN QUESTION 7
}
// FIN QUESTION 4

int main(int argc, char** argv){
  int N;

  double *A;
  double *B;
  double *C;

  double t0 = 0., t1 = 0., duration = 0.;

  N = (argc < 2)?1000:atoi(argv[1]);
  fprintf(stdout, "Matrix Multiplication\n  Size: %dx%d\n", N, N);

  // Memory allocation
  A = (double*) malloc(sizeof(double) * N * N);
  B = (double*) malloc(sizeof(double) * N * N);
  C = (double*) malloc(sizeof(double) * N * N);

  // Value initialization
  init(A, B, C, N);

  // QUESTION 8
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //FIN QUESTION 8

  // QUESTION 1
  double *d_A, *d_B, *d_C;
  hipMalloc(&d_A, sizeof(double) * N * N);
  hipMalloc(&d_B, sizeof(double) * N * N);
  hipMalloc(&d_C, sizeof(double) * N * N);
  // FIN QUESTION 1

  // QUESTION 2
  hipMemcpy(d_A, A, sizeof(double) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, sizeof(double) * N * N, hipMemcpyHostToDevice);
  // FIN QUESTION 2

  // QUESTION 3
  int nbBlocks = N / BLOCK_WIDTH;
  if(N % BLOCK_WIDTH) nbBlocks++;
  dim3 gridSize(nbBlocks, nbBlocks);
  dim3 blockSize(BLOCK_WIDTH, BLOCK_WIDTH);
  // FIN QUESTION 3

  // QUESTION 4
  hipEventRecord(start); // QUESTION 8
  MulMatrixKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
  hipEventRecord(stop); // QUESTION 8
  // FIN QUESTION 4

  // QUESTION 5
  hipMemcpy(C, d_C, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  // FIN QUESTION 5

  // QUESTION 8
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Matrice %dx%d\n\tTemps: %f s\n", N, N, milliseconds/1000);
  // FIN QUESTION 8

  // Compute multiplication
  t0 = get_elapsedtime();
  mult(A, B, C, N);
  t1 = get_elapsedtime();

  // Pretty print
  duration = (t1 - t0);
  uint64_t nb_op = N * N * N;
  fprintf(stdout, "Performance results: \n");
  fprintf(stdout, "  Time: %lf s\n", duration);
  fprintf(stdout, "  MFlops: %.2f\n", (nb_op / duration)*1E-6);

  free(A);
  free(B);
  free(C);

  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}
